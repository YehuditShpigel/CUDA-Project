#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include<memory>
#include<stdexcept>
#include<string>
using namespace std;

__device__ bool isInMat(int x, int y, int width, int height) {
	return x >= 0 && y >= 0 && x < width&& y < height;
}

__device__ int posInMatReflectBorder(int row, int col, int width, int height) {
	int x = col, y = row;
	if (col < 0)
		x = -col;
	if (row < 0)
		y = -row;
	if (col >= width)
		x = width - (col + 2 - width);
	if (row >= height)
		y = height - (row + 2 - height);

	return (y * width + x);
}

__global__ void convolvKernel(float* dst, const float* src, const float* filter, int width, int height, int filterWidth, int filterHeight/*, int x, int y*/)
{

	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int pos = y * width + x;
	bool isInArray = y < height&& x < width;
	float sum = 0;
	if (!isInArray)
		return;
	for (int filter_row = 0, row = (y - filterHeight / 2); filter_row < filterHeight; filter_row++, row++) {
		for (int filter_col = 0, col = x - filterWidth / 2; filter_col < filterWidth; filter_col++, col++) {
			sum += src[posInMatReflectBorder(row, col, width, height)] * filter[filter_row * filterWidth + filter_col];
		}
	}
	dst[pos] = sum;
}

#define ASSERT_CUDA_SUCCESS(cudaStatus,msg){      \
	if (cudaStatus != hipSuccess) {      \
		throw runtime_error(msg);		  \
	}								      \
}

void  convolvWithCuda(float* dst, const float* src, const float* filter, int width, int height, int filterWidth, int filterHeight)
{
	hipError_t cudaStatus;
	size_t size = width * height;

	int threadsPerBlock = 32;
	int numBlocks_x = (width / threadsPerBlock) + 1;
	int numBlocks_y = (height / threadsPerBlock) + 1;

	dim3 threads_per_block_dim(threadsPerBlock, threadsPerBlock);
	dim3 blocks_grid_dim(numBlocks_x, numBlocks_y);

	convolvKernel << <blocks_grid_dim, threads_per_block_dim >> > (dst, src, filter, width, height, filterWidth, filterHeight);

	cudaStatus = hipGetLastError();
	ASSERT_CUDA_SUCCESS(cudaStatus, "addKernel launch failed: " + string(hipGetErrorString(cudaStatus)) + "\n");

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	ASSERT_CUDA_SUCCESS(cudaStatus, "hipDeviceSynchronize returned error code" + to_string(cudaStatus) + "after launching addKernel!\n");
}

